
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "time.h"
#include <iostream>

// Defining Number of elements in Array
#define N	10000000

// Defining vector addition function for CPU
void cpuAdd(int *h_a, int *h_b, int *h_c)
{
	int tid = 0;
	while (tid < N)
	{
		h_c[tid] = h_a[tid] + h_b[tid];
		tid += 1;
	}
}

int main(void)
{
	int *h_a, *h_b, *h_c;
	h_a = (int *)malloc(sizeof(int) * N);
	h_b = (int *)malloc(sizeof(int) * N);
	h_c = (int *)malloc(sizeof(int) * N);
	// Initializing two arrays for addition
	for (int i = 0; i < N; i++)
	{
		h_a[i] = i + 1;
		h_b[i] = i - 1;
	}
	// Start Time
	clock_t start_h = clock();
	// Calling CPU function for vector addition
	cpuAdd(h_a, h_b, h_c);
	// End Time
	clock_t end_h = clock();
	// Printing Answer
	printf("Vector addition on CPU\n");
	// Execute Time
	double time_h = (double)(end_h - start_h) / CLOCKS_PER_SEC;
	printf("N = %d \n", N);
	printf("Execute time: %f seconds \n", time_h);
	
	for (int i = 9999990; i < N; i++)
	{
	printf("The sum of %d element is %d + %d = %d\n", i, h_a[i], h_b[i], h_c[i]);
	}
	
	free(h_a);
	free(h_b);
	free(h_c);
	system("pause");
	return 0;
}