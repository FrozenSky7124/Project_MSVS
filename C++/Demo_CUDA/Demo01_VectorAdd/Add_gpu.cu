#include "hip/hip_runtime.h"
#include "stdio.h"
#include "time.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

// Defining number of elements in Array
#define N 10000000

// Defining Kernel function for vector addition
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c)
{
	// Getting block index of current kernel
	int tid = blockIdx.x * 1000 + threadIdx.x;
	if (tid < N)
		d_c[tid] = d_a[tid] + d_b[tid];
}

int main(void)
{
	// Defining host arrays
	int *h_a, *h_b, *h_c;
	h_a = (int *)malloc(sizeof(int) * N);
	h_b = (int *)malloc(sizeof(int) * N);
	h_c = (int *)malloc(sizeof(int) * N);
	// Defining device pointers
	int *d_a, *d_b, *d_c;
	// Initializing two arrays for addition
	for (int i = 0; i < N; i++)
	{
		h_a[i] = i + 1;
		h_b[i] = i - 1;
	}
	// Allocate the memory
	hipMalloc((void **)&d_a, sizeof(int) * N);
	hipMalloc((void **)&d_b, sizeof(int) * N);
	hipMalloc((void **)&d_c, sizeof(int) * N);
	// Copy input arrays from host to device memory
	hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
	// Start Time
	clock_t start_h = clock();
	// Calling kernels with N blocks and one thread per block, passing device pointers as parameters
	gpuAdd <<<10000, 1000 >>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	// End Time
	clock_t end_h = clock();
	// Copy result back to host memory from device memory
	hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
	printf("Vector addition on GPU \n");
	// Execute Time
	double time_h = (double)(end_h - start_h) / CLOCKS_PER_SEC;
	printf("N = %d \n", N);
	printf("Execute time: %f seconds \n", time_h);
	// Printing result on console
	
	for (int i = 9999990; i < N; i++)
	{
		printf("The sum of %d element is %d + %d = %d\n", i, h_a[i], h_b[i], h_c[i]);
	}
	
	// Free up memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	system("pause");
	return 0;
}
