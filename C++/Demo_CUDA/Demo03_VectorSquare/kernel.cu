#include "stdio.h"
#include<iostream>
#include <hip/hip_runtime.h>

//Defining number of elements in Array
#define N	5
//Kernel function for squaring number
__global__ void gpuSquare(float *d_in, float *d_out) {
	//Getting thread index for current kernel
	int tid = threadIdx.x;	// handle the data at this index
	float temp = d_in[tid];
	d_out[tid] = temp*temp;
}

int main(void) {
	//Defining Arrays for host
	float h_in[N], h_out[N];
	//Defining Pointers for device
	float *d_in, *d_out;

	// allocate the memory on the cpu
	hipMalloc((void**)&d_in, N * sizeof(float));
	hipMalloc((void**)&d_out, N * sizeof(float));
	//Initializing Array
	for (int i = 0; i < N; i++) {
		h_in[i] = i;
	}
	//Copy Array from host to device
	hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
	//Calling square kernel with one block and N threads per block
	gpuSquare<<<1, N>>>(d_in, d_out);
	//Coping result back to host from device memory
	hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
	//Printing result on console
	printf("Square of Number on GPU \n");
	for (int i = 0; i < N; i++) {
		printf("The square of %f is %f\n", h_in[i], h_out[i]);
	}
	//Free up memory
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}